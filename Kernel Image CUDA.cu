#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Kernel.cu
 Author      : Samuele
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#define TILE_WIDTH 16
#define RESULTS_NUM 7
#define KERNEL_SIZE 3
#define INPUT_TILE_WIDTH TILE_WIDTH + KERNEL_SIZE - 1

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
//#include <chrono>
#include "PPM.h"
#include "kernels.h"




static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)



__constant__ float deviceKernels[3*3 * RESULTS_NUM];



__global__ void kernelConvolution(float* img, float** output ,const int imageWidth, const int imageHeight, const int imageChannels){
	__shared__ float imgTile[INPUT_TILE_WIDTH][INPUT_TILE_WIDTH]; //condivisa a livello di blocco


	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;


	if((row < imageHeight) && (col < imageWidth)){
		int kIndex;
		int a,b;
		int border = KERNEL_SIZE / 2;
		int rowImgBorder,colImgBorder,rowTileBorder,colTileBorder,rowInputBorder,colInputBorder;


		int tileRow = row % TILE_WIDTH;
		int tileCol = col % TILE_WIDTH;

		float element;
		float sum[RESULTS_NUM];
		for(int h = 0; h < RESULTS_NUM; h++)
			sum[h] = 0;

		/* offset per gestire i bordi immagine, bordi tile e bordi dell'input */
		rowInputBorder = (tileRow == 0)? border : 0;
		colInputBorder = (tileCol == 0)? border : 0;

		rowTileBorder = (tileRow == TILE_WIDTH - 1)? border : 0;
		colTileBorder = (tileCol == TILE_WIDTH - 1)? border : 0;

		rowImgBorder = (row == imageHeight - 1)? border : 0;
		colImgBorder = (col == imageWidth - 1)? border : 0;



		//printf("row: %i, col: %i,tileCell: %i, tileRow: %i, tileCol: %i\n",row,col,tileCell,tileRow,tileCol);
		for(int c = 0; c < imageChannels; c++){

			/* caricamento del'input tile in shared memory */
			element = img[(row * imageWidth + col) * imageChannels + c];


			/*
			 * devo gestire tre casi principali:
			 *	- bordo immagine: per i pixel necessari per la convoluzione che "vanno fuori" dall'immagine vengono usati i corrispondenti pixel di bordo
			 *	- bordo del tile non di bordo immagine: per i pixel necessari alla convoluzione bisogna prendere pixel che apparterrebbero ad altri tile adiacenti
			 *	- cella non di bordo: ho tutti i pixel necessari alla convoluzione
			 */

			if((row == 0)||(row == imageHeight-1)||(col == 0)||(col == imageWidth-1)){ //bordi immagine

				if(((row == 0)||(row == imageHeight-1))&&((col == 0)||(col == imageWidth-1))){ //angoli immagine
					for(a = 0; a < border+1; a++){
						for(b = 0; b < border+1; b++)
							imgTile[tileRow+a+rowImgBorder][tileCol+b+colImgBorder]=element;
					}
				}

				else if(((row == 0)||(row == imageHeight-1))&&((tileCol == 0)||(tileCol == TILE_WIDTH-1))){ //angoli intermedi superiori e inferiori sinistri
					for(a = 0; a < border+1; a++){
						for(b = 0; b < border+1; b++){
							if(((a < border)&&(row==0))||((a > 0)&&(row==imageHeight-1)))
								imgTile[tileRow+a+rowImgBorder][tileCol+b+colTileBorder]=element;
							else
								imgTile[tileRow+a+rowImgBorder][tileCol+b+colTileBorder]=img[((row-rowInputBorder+a) * imageWidth + (col-colInputBorder+b)) * imageChannels + c];
						}
					}
				}

				else if(((col == 0)||(col == imageWidth-1))&&((tileRow == 0)||(tileRow == TILE_WIDTH-1))){ //angoli intermedi destri e sinistri
					for(a = 0; a < border+1; a++){
						for(b = 0; b < border+1; b++){
							if(((b < border)&&(col==0))||((b > 0)&&(col==imageWidth-1)))
								imgTile[tileRow+a+rowTileBorder][tileCol+b+colImgBorder]=element;
							else
								imgTile[tileRow+a+rowTileBorder][tileCol+b+colImgBorder]=img[((row-rowInputBorder+a) * imageWidth + (col-colInputBorder+b)) * imageChannels + c];
						}
					}
				}

				else{ //lati immagine
					if((row == 0)||(row == imageHeight-1)){ //lati superiore e inferiore
						for(a = 0; a < border+1; a++)
							imgTile[tileRow+a+rowImgBorder][tileCol+border]=element;
					}
					else if((col == 0)||(col == imageWidth-1)){ //lati sinistro e destro
						for(a = 0; a < border+1; a++)
							imgTile[tileRow+border][tileCol+a+colImgBorder]=element;
					}
				}
			}


			else if((tileRow == 0)||(tileRow == TILE_WIDTH-1)||(tileCol == 0)||(tileCol == TILE_WIDTH-1)){ //bordi tile interni

				if(((tileRow == 0)||(tileRow == TILE_WIDTH-1))&&((tileCol == 0)||(tileCol == TILE_WIDTH-1))){ //angoli tile interni
					for(a = 0; a < border+1; a++){
						for(b = 0; b < border+1; b++)
							imgTile[tileRow+a+rowTileBorder][tileCol+b+colTileBorder]=img[((row-rowInputBorder+a) * imageWidth + (col-colInputBorder+b)) * imageChannels + c];
					}

				}
				else{ //bordi tile interni
					if((tileRow == 0)||(tileRow == TILE_WIDTH-1)){
						for(a = 0; a < border+1; a++)
							imgTile[tileRow+a+rowTileBorder][tileCol+border]=img[((row-rowInputBorder+a) * imageWidth + col) * imageChannels + c];
					}
					else if((tileCol == 0)||(tileCol == TILE_WIDTH-1)){
						for(a = 0; a < border+1; a++)
							imgTile[tileRow+border][tileCol+a+colTileBorder]=img[(row * imageWidth + (col-colInputBorder+a)) * imageChannels + c];
					}
				}

			}


			else //cella non di bordo
				imgTile[tileRow+border][tileCol+border] = element;






			__syncthreads();


			/*calcolo dell'output*/
			for(int i = 0; i < KERNEL_SIZE; i++){
				//yIndex = ((row - 1 + i) < 0) ? 0 : ((row - 1 + i) >= imageHeight) ? imageHeight - 1 : row - 1 + i;
				for(int j = 0; j < KERNEL_SIZE; j++){
					//xIndex = ((col - 1 + j) < 0) ? 0 : ((col - 1 + j) >= imageWidth) ? imageWidth - 1 : col - 1 + j;
					kIndex = (2 - i)*3 + (2 -j);

					for(int h = 0; h < RESULTS_NUM; h++)
						//sum[h] += img[(yIndex * imageWidth + xIndex) * imageChannels + c] * deviceKernels[kIndex + 3*3*h];
						sum[h] += imgTile[tileRow + i][tileCol + j] * deviceKernels[kIndex + 3*3*h];
				}
			}
			for(int x = 0; x < RESULTS_NUM; x++){
				//output[x][(row * imageWidth + col) * imageChannels + c] = imgTile[tileRow][tileCol];
				output[x][(row * imageWidth + col) * imageChannels + c] = sum[x];
				sum[x] = 0;
			}
			__syncthreads();
		}
	}

}


int main(int argc,  char** argv){

    Image_t* inputImage = PPM_import(argv[1]);

    const int imageWidth = Image_getWidth(inputImage);
    const int imageHeight = Image_getHeight(inputImage);
    const int imageChannels = Image_getChannels(inputImage);
    const int imageDataSize = sizeof(float)*imageWidth*imageHeight*imageChannels;
    int i;

    float *hostInput;
    float *deviceInput;
    float **deviceOutput;
    float *hostOutput[RESULTS_NUM];
    float *hostKernels[RESULTS_NUM];



    hostInput = Image_getData(inputImage);

    //creo le immagini di output
    Image_t* outputImages[RESULTS_NUM];

    std::cout << imageWidth << "x" << imageHeight << "-" << imageChannels << std::endl;

    //copio in constant memory i dati delle matrici kernel
    hostKernels[0] = identity;
    hostKernels[1] = edge1;
    hostKernels[2] = edge2;
    hostKernels[3] = edge3;
    hostKernels[4] = sharpen;
    hostKernels[5] = boxblur;
    hostKernels[6] = gaussianblur3;

    //alloco la memoria (global) per contenere i dati dell'immagine e ci copio i dati
    CUDA_CHECK_RETURN(hipMalloc((void**)&deviceInput,imageDataSize));
    CUDA_CHECK_RETURN(hipMemcpy(deviceInput,hostInput,imageDataSize,hipMemcpyHostToDevice));

    float *devicePointers[RESULTS_NUM]; //array di puntatori a locazioni di memoria device
    for(i = 0; i < RESULTS_NUM; i++){
    	outputImages[i] = Image_new(imageWidth,imageHeight,imageChannels);
    	hostOutput[i] = Image_getData(outputImages[i]);
    	CUDA_CHECK_RETURN(hipMalloc((void**)&devicePointers[i],imageDataSize)); //alloco le varie locazioni di memoria device per le immagini e salvo il puntatore nell'array
    	CUDA_CHECK_RETURN(hipMemcpy(devicePointers[i],hostOutput[i],imageDataSize,hipMemcpyHostToDevice)); //trasferisco i dati immagini nelle locazioni appena create
    	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(deviceKernels), hostKernels[i],sizeof(float)*3*3,sizeof(float)*3*3*i)); //trasferisco i dati dei kernel in constant memory
    }
    CUDA_CHECK_RETURN(hipMalloc((void**)&deviceOutput,sizeof(float*) * RESULTS_NUM)); //alloco memoria device per ospitare l'array di locazioni di memoria device
    CUDA_CHECK_RETURN(hipMemcpy(deviceOutput,devicePointers,sizeof(float*) * RESULTS_NUM,hipMemcpyHostToDevice));

    dim3 gridDim(ceil((float)imageWidth / TILE_WIDTH),ceil((float)imageHeight / TILE_WIDTH));
    dim3 blockDim(TILE_WIDTH,TILE_WIDTH);
    //printf("%i,%i\n",(imageWidth / TILE_WIDTH) + 1,(imageHeight / TILE_WIDTH) + 1);

    //auto start = std::chrono::system_clock::now();

    kernelConvolution<<<gridDim,blockDim>>>(deviceInput,deviceOutput,imageWidth,imageHeight,imageChannels);

    hipDeviceSynchronize();

    //copio indietro i risultati e imposto i dati immagine
    for(i = 0; i < RESULTS_NUM; i++){
    	CUDA_CHECK_RETURN(hipMemcpy(hostOutput[i],devicePointers[i],imageDataSize,hipMemcpyDeviceToHost));
    	Image_setData(outputImages[i],hostOutput[i]);
    }


    //esporto le immagini
	//PPM_export("processed/identity.ppm",outputImages[0]);
    //PPM_export("processed/edge1.ppm",outputImages[1]);
   // PPM_export("processed/edge2.ppm",outputImages[2]);
    //PPM_export("processed/edge3.ppm",outputImages[3]);
    //PPM_export("processed/sharpen.ppm",outputImages[4]);
    PPM_export("processed/boxblur.ppm",outputImages[5]);
    //PPM_export("processed/gaussianblur3.ppm",outputImages[6]);

    //libero la memoria
    CUDA_CHECK_RETURN(hipFree(deviceInput));
    for(int i = 0; i < RESULTS_NUM; i++)
    	CUDA_CHECK_RETURN(hipFree(devicePointers[i]));
    CUDA_CHECK_RETURN(hipFree(deviceOutput));





    //auto end = std::chrono::system_clock::now();
    //std::chrono::duration<double> elapsed = end-start;

    //printf("Elapsed time:%d\n",elapsed);





	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

