#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : Kernel.cu
 Author      : Samuele
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#define TILE_WIDTH 16
#define MAX_KERNEL 7


#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <chrono>
#include "PPM.h"
#include "kernels.h"




static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)



__constant__ float deviceKernel[7*7];



__global__ void kernelConvolution(float* img, float* output ,const int imageWidth, const int imageHeight, const int imageChannels, const int KERNEL_SIZE){

	const int INPUT_TILE_WIDTH = TILE_WIDTH + MAX_KERNEL - 1;
	__shared__ float imgTile[INPUT_TILE_WIDTH][INPUT_TILE_WIDTH]; //condivisa a livello di blocco

	//printf("%i-%i\n",blockDim.x,blockDim.y);
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	int border = KERNEL_SIZE / 2;

	int kIndex;
	float sum = 0;

	//in quale INPUT_TILE sono (e quindi anche in quale TILE)
	int h = blockIdx.x;
	int w = blockIdx.y;

	//dove sono all'interno del mio INPUT_TILE
	int tileRow = threadIdx.y;
	int tileCol = threadIdx.x;
	//int tileChannel = threadIdx.z;

	//in quale rispettivo pixel di immagine devo lavorare
	int wOff = 	(row <= border)? 0 : tileRow - border;
	int hOff = 	(col <= border)? 0 : tileCol - border;

	int imgRow = ((w * TILE_WIDTH + wOff) < imageHeight)? w * TILE_WIDTH + wOff : imageHeight - 1;
	int imgCol = ((h * TILE_WIDTH + hOff) < imageWidth)? h * TILE_WIDTH + hOff : imageWidth - 1;


	for(int c = 0; c < imageChannels; c++){
		imgTile[tileRow][tileCol] = img[(imgRow * imageWidth + imgCol)*imageChannels + c];


		__syncthreads();


		if((tileRow >= border)&&(tileRow < TILE_WIDTH + border)&&(tileCol >= border)&&(tileCol < TILE_WIDTH + border)&&((w * TILE_WIDTH + wOff)<imageHeight)&&((h * TILE_WIDTH + hOff)<imageWidth)){
			for(int i = 0; i < KERNEL_SIZE; i++){
				for(int j = 0; j < KERNEL_SIZE; j++){
					kIndex = (KERNEL_SIZE-1 - i) * KERNEL_SIZE + (KERNEL_SIZE-1 -j);
					sum += imgTile[tileRow + i - border][tileCol + j - border] * deviceKernel[kIndex];
				}
			}
			output[(imgRow * imageWidth + imgCol) * imageChannels + c] = sum;//imgTile[tileRow-1][tileCol-1];
			sum = 0;
		}

		__syncthreads();

	}

}


int main(int argc,  char** argv){

    Image_t* inputImage = PPM_import(argv[1]);

    const int KERNEL_SIZE = (*argv[2] != '3' && *argv[2] != '5' && *argv[2] != '7')? 3 : (int)*argv[2] - '0';
    const int imageWidth = Image_getWidth(inputImage);
    const int imageHeight = Image_getHeight(inputImage);
    const int imageChannels = Image_getChannels(inputImage);
    const int imageDataSize = sizeof(float)*imageWidth*imageHeight*imageChannels;

    Image_t* outputImage = Image_new(imageWidth,imageHeight,imageChannels);

    float *hostInput = Image_getData(inputImage);;
    float *hostOutput = Image_getData(outputImage);
    float *deviceInput;
    float *deviceOutput;

    float* hostKernel = (*argv[2] == '5')? kernel5 : (*argv[2] == '7')? kernel7 : kernel3;



    //alloco la memoria (global) per contenere i dati dell'immagine e ci copio i dati
    CUDA_CHECK_RETURN(hipMalloc((void**)&deviceInput,imageDataSize));
    CUDA_CHECK_RETURN(hipMemcpy(deviceInput,hostInput,imageDataSize,hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipMalloc((void**)&deviceOutput,imageDataSize)); //alloco le varie locazioni di memoria device per le immagini e salvo il puntatore nell'array
	CUDA_CHECK_RETURN(hipMemcpy(deviceOutput,hostOutput,imageDataSize,hipMemcpyHostToDevice)); //trasferisco i dati immagini nelle locazioni appena create

	CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(deviceKernel), hostKernel,sizeof(float)*KERNEL_SIZE*KERNEL_SIZE)); //trasferisco i dati dei kernel in constant memory


    dim3 gridDim(ceil((float)imageWidth / TILE_WIDTH),ceil((float)imageHeight / TILE_WIDTH));
    dim3 blockDim(TILE_WIDTH + KERNEL_SIZE - 1,TILE_WIDTH + KERNEL_SIZE - 1);
    //printf("%i,%i\n%i,%i\n",gridDim.x,gridDim.y,blockDim.x,blockDim.y);

    auto start = std::chrono::system_clock::now();

    kernelConvolution<<<gridDim,blockDim>>>(deviceInput,deviceOutput,imageWidth,imageHeight,imageChannels,KERNEL_SIZE);

    hipDeviceSynchronize();

    auto end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed = end-start;
    std::cout << elapsed.count();

    //copio indietro i risultati e imposto i dati immagine
	CUDA_CHECK_RETURN(hipMemcpy(hostOutput,deviceOutput,imageDataSize,hipMemcpyDeviceToHost));

	Image_setData(outputImage,hostOutput);

    PPM_export("/home/samuele/Documenti/Università/Parallel Computing/Progetto finale/Kernel image/outputs/output_CUDA.ppm",outputImage);

    //libero la memoria
    CUDA_CHECK_RETURN(hipFree(deviceInput));
    CUDA_CHECK_RETURN(hipFree(deviceOutput));


	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

